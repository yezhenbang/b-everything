#include "hip/hip_runtime.h"
/*
 * Author: Rachel
 * <zhangruiqing01@baidu.com>
 *
 * File: bitonic_sort.cu
 * Create Date: 2015-08-05 17:10:44
 *
 */

#include "gputimer.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define NThreads 8
#define NBlocks 4

#define Num NThreads* NBlocks


/* 
Align2(n)
Align2(k)

block: n/k
threads: k/2

total threads: n / 2
one thread complexity: 
    part1 generate k-sequences: log(k)*log(k)  约10算术/位运算操作+1比较分支+1交换
    part2 loop reduce N to k: (log(N)-log(k))*log(k)  约10算术/位运算操作+1比较分支+1交换
*/
__device__ void bitonic_sort_k(float* data, int data_num, int k) {
    int bid = BlockID(), tid = ThreadID();

    ShareMemory float share_data[];
    share_data[tid] = data[tid];
    share_data[tid + k*2] = data[tid + k*2];

    // generate (n/k) bitonic sequences of length 2k
    for (int half_bitonic_seq_size = 1; half_bitonic_seq_size <= k; half_bitonic_seq_size<<1) {
        int bitonic_seq_size = half_bitonic_seq_size << 1;
        for (int inc = half_bitonic_seq_size; inc > 0; inc >>= 1) {
            int low = (tid << 1) - (tid & inc - 1);
            bool reverse = bitonic_seq_size & low == 0;
            bool to_swap = share_data[low] < share_data[low + inc];
            // if (to_swap ^ reverse) {
            //     swap(share_data[low], share_data[low + inc]);
            // }
            // Using max min might be faster than swap???
            float tmp = share_data[low];
            share_data[low] = max(tmp, share_data[low + inc]);
            share_data[low + inc] = min(tmp, share_data[low+inc]);
            __syncthreads();
        }
    }

    // done , got (n/k) bitonic sequences of length 2k
    // [\\\k\\\///k/// | \\\k\\\///k/// \\\k\\\///k/// \\\k\\\///k///]
    int low = (tid << 1) - (tid & k - 1);
    share_data[low] = max(share_data[low], share_data[low + k]);
    // drop lower part, reduce data size to n/2 
    // [\\\k///       \\\k///       \\\k///       \\\k///       ] keep higher part

    // [       \\\k///       \\\k///       \\\k///       \\\k///] drop lower part

    __syncthreads();

    // ===============================

    int drop_tid = 1 << 1; // half threads done
    int remain_datasize = data_num >> 1;

    while (remain_datasize > k && drop_tid & tid == 0) {
        // loop merge 2k and reduce k, until last k
        int half_bitonic_seq_size = k; // constraint k
        int drop_offset = (drop_tid << 1 - 1) * k;
        for (int inc = half_bitonic_seq_size; inc > 0; inc >>= 1) {
            int low = (tid << 1) - (tid & inc - 1);
            bool reverse = bitonic_seq_size & low == 0;
            bool to_swap = share_data[low] < share_data[low + inc + drop_offset];
            if (to_swap ^ reverse) {
                swap(share_data[low], share_data[low + inc + drop_offset]);
            }
            __syncthreads();
        }

        remain_datasize >> 1;
        drop_tid << 1;

    } while (remain_datasize > k && drop_tid & tid == 0);

    // remain tid 0~k-1
    data[tid] = share_data[tid];
}



using namespace Gadgetron;

__device__ void swap(int& a, int& b) {
    int t = a;
    a = b;
    b = t;
}

__global__ void bitonic_sort(int* arr) {
    extern __shared__ int shared_arr[];
    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // const unsigned int tid = threadIdx.x;
    shared_arr[tid] = arr[tid];
    __syncthreads();

    // for(int i=2; i<=blociDim.x; i<<=1){
    for (unsigned int i = 2; i <= Num; i <<= 1) {
        for (unsigned int j = i >> 1; j > 0; j >>= 1) {
            unsigned int tid_comp = tid ^ j;
            if (tid_comp > tid) {
                if ((tid & i) == 0) { // ascending
                    if (shared_arr[tid] > shared_arr[tid_comp]) {
                        swap(shared_arr[tid], shared_arr[tid_comp]);
                    }
                } else { // desending
                    if (shared_arr[tid] < shared_arr[tid_comp]) {
                        swap(shared_arr[tid], shared_arr[tid_comp]);
                    }
                }
            }
            __syncthreads();
        }
    }
    arr[tid] = shared_arr[tid];
}

int main(int argc, char* argv[]) {
    GPUTimer timer;
    int* arr = ( int* )malloc(Num * sizeof(int));

    // init array value
    time_t t;
    srand(( unsigned )time(&t));
    for (int i = 0; i < Num; i++) {
        arr[i] = rand() % 1000;
    }

    // init device variable
    int* ptr;
    hipMalloc(( void** )&ptr, Num * sizeof(int));
    hipMemcpy(ptr, arr, Num * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < Num; i++) {
        printf("%d\t", arr[i]);
    }
    printf("\n");

    dim3 blocks(NBlocks, 1);
    dim3 threads(NThreads, 1);

    timer.start();
    bitonic_sort< < < blocks, threads, Num * sizeof(int) > > >(ptr);
    // bitonic_sort<<<1,Num,Num*sizeof(int)>>>(ptr);
    timer.stop();

    hipMemcpy(arr, ptr, Num * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < Num; i++) {
        printf("%d\t", arr[i]);
    }
    printf("\n");

    hipFree(ptr);
    return 0;
}